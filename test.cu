#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

/*
The header /usr/local/cuda-12.1/include/hipcub/hipcub.hpp indirectly includes the
header /usr/local/cuda-12.1/include/cuda/std/detail/__config, that defines
_Float16 as

#define _Float16 __half

This conflicts with GCC's use of _Float16 on ARM and on x86 systems with SSE2,
according to ISO/IEC TS 18661-3:2015.

_Float16 is used in GCC's headers avx512fp16vlintrin.h and avx512fp16intrin.h,
that are included by <immintrin.h>

On those systems, including <immintrin.h> after <hipcub/hipcub.hpp> causes a syntax
error at compilation.

Uncomment the next line to work around the issue.
*/

//#undef _Float16

#include <immintrin.h>

__global__
void unused() {}
